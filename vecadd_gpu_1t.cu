//Nilofar Ali
//CSC 656 CP 4
// May 16, 2024


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void __global__add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1 << 29;
  float *x, *y;

  // Allocate Managed Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // Initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  __global__add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}